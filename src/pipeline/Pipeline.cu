#include "hip/hip_runtime.h"
#include "Pipeline.cuh"

#include <atomic>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <mutex>
#include <condition_variable>

condition_variable cv;
atomic<int> nbFinished(0);
mutex global_mutex;

Pipeline::Pipeline(Raster& rasterIn, Raster* rasterOut, const LightingParams lightParams, uint tileSize, uint tileBuffer, float exaggeration, uint startTile, GDALDataType outputType): 
rasterIn(rasterIn), rasterOut(rasterOut){
    const uint nbTiles = (uint)ceil((float)rasterIn.getHeight()/tileSize) * (uint)ceil((float)rasterIn.getWidth()/tileSize);

    for(uint i=0; i<NB_PIPELINE_STAGES; i++){
        stages[i].state = new PipelineState();
        stages[i].id = i;
    }

    stages[0].thread = new thread(Pipeline::readData,  &stages[0], &rasterIn, tileSize, tileBuffer, startTile);
    stages[1].thread = new thread(Pipeline::initTile,  &stages[1], rasterIn.getPixelSize(), exaggeration);
    stages[2].thread = new thread(Pipeline::trace,     &stages[2], lightParams);
    stages[3].thread = new thread(Pipeline::writeData, &stages[3], rasterOut, &rasterIn, outputType);
}

Pipeline::~Pipeline(){
    debug_print("Stopping pipeline \n");
    for(uint i=0; i<NB_PIPELINE_STAGES; i++){
        stages[i].thread->join();
        delete stages[i].state;
        delete stages[i].thread;
    }
}

bool Pipeline::step(){
    while(nbFinished < NB_PIPELINE_STAGES){}
    lock_guard<mutex> lock(global_mutex);
    debug_print("Next pipeline step - Releasing threads \n");

    PipelineState* state1Tmp = stages[0].state;
    stages[0].state = stages[3].state;
    stages[3].state = stages[2].state;
    stages[2].state = stages[1].state;
    stages[1].state = state1Tmp;

    nbFinished = 0;
    for(uint i=0; i < NB_PIPELINE_STAGES; i++){
        stages[i].ready = true;
    }
    cv.notify_all();
    return !stages[3].state->finished;
}

void Pipeline::waitForNextStep(PipelineStage* stage, timePoint startTime){
    int elapsedTime = chrono::duration_cast<chrono::milliseconds>(chrono::high_resolution_clock::now() - startTime).count();
    debug_print("Thread " + STAGE_NAMES[stage->id] + " waiting [" + to_string(elapsedTime) + " ms]\n");
    unique_lock<mutex> lock(global_mutex);
    nbFinished++;
    cv.wait(lock, [stage]{ return stage->ready; });
    //debug_print("Thread " + to_string(stage->id) + " released \n");
    stage->ready = false;
}


void Pipeline::readData(PipelineStage* stage, const Raster* rasterIn, uint tileSize, uint tileBuffer, uint startTile){
    const float noDataValue = rasterIn->getNoDataValue();
    const uint nbTiles = (uint)ceil((float)rasterIn->getHeight()/tileSize) * (uint)ceil((float)rasterIn->getWidth()/tileSize);
    uint nbTileProcessed = 0;
    timePoint startTime = chrono::high_resolution_clock::now();
    for(uint y=0; y<rasterIn->getHeight(); y+=tileSize){
        for(uint x=0; x<rasterIn->getWidth(); x+=tileSize){

            if (nbTileProcessed < startTile){
                nbTileProcessed++;
                continue;
            }

            Pipeline::waitForNextStep(stage, startTime);
            startTime = chrono::high_resolution_clock::now();
            PipelineState* state = stage->state;
            print_atomic("Processing tile " + to_string(nbTileProcessed) + "/" + to_string(nbTiles) + " (" + to_string(100*nbTileProcessed/nbTiles) + "%)...\n");

            state->id = nbTileProcessed;
            state->x = x;
            state->y = y;
            state->width  = min(tileSize, rasterIn->getWidth()-x);
            state->height = min(tileSize, rasterIn->getHeight()-y);
            state->extent.xMin = max(0, (int)x-(int)tileBuffer);
            state->extent.yMin = max(0, (int)y-(int)tileBuffer);
            state->extent.xMax = min(rasterIn->getWidth(), x+state->width+tileBuffer);
            state->extent.yMax = min(rasterIn->getHeight(),y+state->height+tileBuffer);

            if(state->dataIn != nullptr){
                delete state->dataIn;
            }
            if(state->dataOut != nullptr){
                delete state->dataOut; 
            }
            state->dataIn = new Array2D<float>(state->extent.xMax-state->extent.xMin, state->extent.yMax-state->extent.yMin);
            state->dataOut = new Array2D<float>(state->extent.xMax-state->extent.xMin, state->extent.yMax-state->extent.yMin);

            rasterIn->readData(state->dataIn->begin(), state->extent.xMin, state->extent.yMin, state->extent.xMax-state->extent.xMin, state->extent.yMax-state->extent.yMin);

            uint i=0;
            state->hasData = false;
            int xmax = state->x+(int)state->width;
            int ymax = state->y+(int)state->height;
            for(int curY = state->extent.yMin; curY < state->extent.yMax; curY++){
                for(int curX = state->extent.xMin; curX < state->extent.xMax; curX++){
                    if( curY >= state->y && curY < ymax && curX >= state->x && curX < xmax && (*state->dataIn)[i] != noDataValue ){
                        state->hasData = true;
                    }
                    (*state->dataOut)[i] = (*state->dataIn)[i];
                    i++;
                }
            }

            nbTileProcessed++;
        }
    }
    debug_print("> Thread " + STAGE_NAMES[stage->id] + " finished...\n");
    for(uint i=3; i>0; i--){
        PipelineState* state = stage->state;
        state->finished = true;
        Pipeline::waitForNextStep(stage, startTime);
    }
    debug_print("> Thread " + STAGE_NAMES[stage->id] + " exit\n");
}

void Pipeline::initTile(PipelineStage* stage, float pixelSize, float exaggeration){
    PipelineState* state = stage->state;
    timePoint startTime = chrono::high_resolution_clock::now();
    while(!state->finished){
        Pipeline::waitForNextStep(stage, startTime);
        startTime = chrono::high_resolution_clock::now();
        state = stage->state;
        if(state->hasData && state->id >= 0){
            debug_print("> Building tree for tile " + to_string(state->id+1) + "...\n");
            if(state->tracer != nullptr){
                delete state->tracer;
            }
            state->tracer = new Tracer(*state->dataOut, pixelSize, exaggeration);
            state->tracer->init(false);
        }else if(state->id >= 0){
            logger::cout() << "> Tile "<< state->id+1  <<" skipped because it had no data \n";
        }
    }
    debug_print("> Thread " + STAGE_NAMES[stage->id] + " finished...\n");
    Pipeline::waitForNextStep(stage, startTime);
    Pipeline::waitForNextStep(stage, startTime);
    debug_print("> Thread " + STAGE_NAMES[stage->id] + " exit\n");
}

void Pipeline::trace(PipelineStage* stage, const LightingParams params){
    PipelineState* state = stage->state;
    timePoint startTime = chrono::high_resolution_clock::now();
    while(!state->finished){
        Pipeline::waitForNextStep(stage, startTime);
        startTime = chrono::high_resolution_clock::now();
        state = stage->state;
        if(state->hasData && state->id >= 0){
            debug_print("> Tracing tile " + to_string(state->id+1) + "...\n");
            state->tracer->trace(true, params);
        }
    }
    debug_print("> Thread " + STAGE_NAMES[stage->id] + " finished...\n");
    Pipeline::waitForNextStep(stage, startTime);
    debug_print("> Thread " + STAGE_NAMES[stage->id] + " exit\n");
}

void Pipeline::writeData(PipelineStage* stage, const Raster* const rasterOut, const Raster* const rasterIn, GDALDataType outputType){
    PipelineState* state = stage->state;
    timePoint startTime = chrono::high_resolution_clock::now();
    const float noDataValue = rasterIn->getNoDataValue();
    while(!state->finished){
        Pipeline::waitForNextStep(stage, startTime);
        startTime = chrono::high_resolution_clock::now();
        state = stage->state;
        if(state->hasData && state->id >= 0){
            debug_print("> Writing tile " + to_string(state->id+1) + "...\n");
            Array2D<float> dataCropped(state->width, state->height);
            uint i=0, j=0;
            for(int curY=state->extent.yMin; curY < state->extent.yMax; curY++){
                for(int curX=state->extent.xMin; curX < state->extent.xMax; curX++){
                    if(curY>=state->y && curY < state->y+(int)state->height && curX>=state->x && curX < state->x+(int)state->width){
                        dataCropped[i++] = (*state->dataIn)[j] == noDataValue ? noDataValue : (*state->dataOut)[j];
                    }
                    j++;
                }
            }

            if(rasterOut != nullptr){
                rasterOut->writeData(dataCropped.begin(), state->x, state->y, state->width, state->height);
            }else{
                const int tileX = state->x / state->width;
                const int tileY = state->y / state->height;
                std::ostringstream oss;
                oss << "./output_tiles/" << std::setw(8) << std::setfill('0') << state->id <<"_tile_" 
                    << std::setw(5) << std::setfill('0') << tileX << "_" << std::setw(5) << std::setfill('0') << tileY << ".tif";
                if(outputType == GDT_Float32){
                    Raster::writeTile(dataCropped.begin(), state->x, state->y, state->width, state->height, rasterIn, oss.str().c_str(), outputType);
                }else if(outputType == GDT_Byte){
                    Array2D<unsigned char> dataCroppedByte(state->width, state->height);
                    for(uint i=0; i<dataCropped.size(); i++){
                        dataCroppedByte[i] = (unsigned char)(255.0f * std::max(std::min(dataCropped[i], 1.0f), 0.0f));
                    }
                    Raster::writeTile(dataCroppedByte.begin(), state->x, state->y, state->width, state->height, rasterIn, oss.str().c_str(), outputType);
                }
                
            }
        }
    }
    debug_print("> Thread " + STAGE_NAMES[stage->id] + " finished...\n");
    debug_print("> Thread " + STAGE_NAMES[stage->id] + " exit\n");
}